
#include <hip/hip_runtime.h>
__host__ __device__ void swap(int* a, int* b){
    int temp = *a;
    *a = *b;
    *b = temp;
}

__host__ __device__ void swap4(int* a, int* b){
    for (int i=0; i<4; ++i)
        swap(a+i, b+i);
};

__host__ __device__ void erase(int* a){
    for (int i=0; i<4; ++i)
        a[i] = 0;
};

// Rewrites 3 memory nodes based on interaction rules
__host__ __device__ void rewrite(int* a, int* b, int* c){
    // Full block
    if (!(a[0] || b[0] || c[0]) || (a[0] && b[0] && c[0]))
        return;

    // Move
    if (!(a[0] || b[0]) || (a[0] && b[0]))
        swap4(&b[0], &c[0]);

    // Duplicate
    if (a[0] < 0 && b[0] == 0)
        swap(&a[1], &a[2]),
        b[0] = -a[0], b[1] = a[3], a[0] *= -1,
        b[2] = a[2] + (a[2]>0?2:-1),
        a[3] = a[2] + (a[2]>0?1:-2),
        b[3] = a[2] + (a[2]>0?3:-3);

    // Nothing to do
    if (!(a[0]>0 && !b[0] && c[0]>0))
        return;

    // Passthrough
    if (   (a[1] >  0 && c[1]  < 0)
        || (a[1] >  0 && a[0] == 1)
        || (c[0] == 1 && c[1]  < 0)){
        swap4(a, c);
        for (int i=1; i<=3; ++i)
            a[i] *= a[i]==-c[1] || a[i]==-c[2] || a[i]==-c[3] ? -1 : 1;
        for (int i=1; i<=3; ++i)
            c[i] *= c[i]== a[1] || c[i]== a[2] || c[i]== a[3] ? -1 : 1;
    };

    // Substitute
    for (int n=0; n<2; ++n){
        int* x = n ? c : a;
        int* y = n ? a : c;
        for (int i=1; i<=3; ++i){
            if (x[0] == 1 && x[1] == -y[i]){
                y[i] = x[2];
                erase(x);
                if (y[2] == -y[3] && y[2] < 0)
                    swap(&y[2], &y[3]);
                if (y[1] == -y[2])
                    erase(y);
            };
        };
    };

    // React
    if (a[0] > 1 && c[0] > 1 && a[1] == -c[1]){
        if (a[0] == c[0])
            a[0] = 1, a[1] = a[2], a[2] = c[2],
            c[0] = 1, c[1] = a[3], c[2] = c[3],
            a[3] = 0, c[3] = 0;
        else
            swap(a, c),
            a[0] *= -1,
            c[0] *= -1,
            a[1] = (a[1]*1103515245+12345)&0x7fffffff, // PRNG
            c[1] = -a[1];
    };
};

// Debug pretty print of a slice of the memory state
void print(int *S, int len){
    for (int j=0; j<4; ++j){
        for (int i=j; i<len*4; i+=4){
                 if (!j && S[i]<-1) printf("DD|");
            else if (!j && S[i]==1) printf("%%%%|");
            else if (S[i]>0) printf("%02x>", S[i]%256);
            else if (S[i]<0) printf("%02x<", (-S[i])%256);
            else if (j<3) printf("  |");
            else printf("__|");
        };
        printf("\n");
    };
};
