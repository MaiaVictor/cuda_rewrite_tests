#include "hip/hip_runtime.h"
#include <stdio.h>
#include "test_inputs.cu"
#include "rewriter.cu"

__global__ void process(int *mem, int len, int j){
    int i = (blockIdx.x*blockDim.x + threadIdx.x)*3 + j;
    if (i >= len-2) return;
    rewrite(mem+i*4, mem+i*4+4, mem+i*4+8);
};

int main(){
    for (int i=0; i < memory_ints; ++i)
        memory[i] = i < program_nodes*4 ? program[i] : 0;

    int *device_memory;
    hipMalloc((void**)&device_memory, sizeof memory);
    hipMemcpy(device_memory, memory, memory_size, hipMemcpyHostToDevice);

    dim3 block_size(16, 1);
    dim3 grid_size(memory_nodes / block_size.x / 3, 1);

    print(memory, 40);
    for (int k=0; k<clocks; ++k)
        for (int j=0; j<3; ++j)
            process<<<block_size, grid_size>>>(device_memory, memory_nodes, j);

    hipMemcpy(memory, device_memory, memory_size, hipMemcpyDeviceToHost);
    hipFree(device_memory);

    print(memory, 40);
}
